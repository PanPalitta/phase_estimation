#include <iostream>
#include <map>
#include <vector>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "rng.h"

using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
            cout << "CUDA call error at" << __FILE__<< ":" << __LINE__ << endl;\
            }} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
            cout << "CURAND call error at" << __FILE__<< ":" << __LINE__ << endl;\
            exit(-1);\
            }} while(0)

/// Note that this function was lifted from http://code.google.com/p/gpmr/
void setDevice(int commRank, int commSize) {
    int devCount;
    int deviceNum = 0;
    CUDA_CALL(hipGetDeviceCount(&devCount));
    FILE * fp = popen("/bin/hostname", "r");
    char buf[1024];
    if (fgets(buf, 1023, fp) == NULL) strcpy(buf, "localhost");
    pclose(fp);
    string host = buf;
    host = host.substr(0, host.size() - 1);
    strcpy(buf, host.c_str());
    if (commRank == 0) {
        map<string, vector<int> > hosts;
        map<string, int> devCounts;
        hosts[buf].push_back(0);
        devCounts[buf] = devCount;

        MPI_Status stat;
        MPI_Request req;
        for (int i = 1; i < commSize; ++i) {
            MPI_Recv(buf, 1024, MPI_CHAR, i, 0, MPI_COMM_WORLD, &stat);
            MPI_Recv(&devCount, 1, MPI_INT, i, 0, MPI_COMM_WORLD, &stat);

            // check to make sure each process on each node reports the same number of devices.
            hosts[buf].push_back(i);
            if (devCounts.find(buf) != devCounts.end()) {
                if (devCounts[buf] != devCount) {
                    printf("Error, device count mismatch %d != %d on %s\n", devCounts[buf], devCount, buf);
                    fflush(stdout);
                    }
                }
            else devCounts[buf] = devCount;
            }
        // check to make sure that we don't have more jobs on a node than we have GPUs.
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            if (it->second.size() > static_cast<unsigned int>(devCounts[it->first])) {
                printf("Error, more jobs running on '%s' than devices - %d jobs > %d devices.\n",
                       it->first.c_str(), static_cast<int>(it->second.size()), devCounts[it->first]);
                fflush(stdout);
                exit(1);
                }
            }

        // send out the device number for each process to use.
        MPI_Irecv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &req);
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            for (unsigned int i = 0; i < it->second.size(); ++i) {
                int devID = i;
                MPI_Send(&devID, 1, MPI_INT, it->second[i], 0, MPI_COMM_WORLD);
                }
            }
        MPI_Wait(&req, &stat);
        }
    else {
        // send out the hostname and device count for your local node, then get back the device number you should use.
        MPI_Status stat;
        MPI_Send(buf, strlen(buf) + 1, MPI_CHAR, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&devCount, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
        MPI_Recv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &stat);
        }
    MPI_Barrier(MPI_COMM_WORLD);
    CUDA_CALL(hipSetDevice(deviceNum));
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    cout <<  device << " " << devProp.name << " Compute Capability: " << devProp.major << "." << devProp.minor << "\n";
}

RngGpu::RngGpu(bool _gaussian, int _n_random_numbers, int seed, int rank):
    n_random_numbers(_n_random_numbers), RngVectorized(_gaussian) {
    int nb_proc;
    MPI_Comm_size(MPI_COMM_WORLD, &nb_proc);
    setDevice(rank, nb_proc);
    CUDA_CALL(hipHostMalloc((void **)&random_numbers,
                             n_random_numbers * sizeof(double)));
    index_random_numbers = n_random_numbers;
    CUDA_CALL(hipMalloc((void **)&dev_random_numbers,
                         n_random_numbers * sizeof(double)));
    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
    if (gaussian) {
        CURAND_CALL(hiprandGenerateNormalDouble(gen, dev_random_numbers,
                                               n_random_numbers, 0.0, 1.0));
    } else {
        CURAND_CALL(hiprandGenerateUniformDouble(gen, dev_random_numbers,
                                                n_random_numbers));
    }
    CUDA_CALL(hipMemcpyAsync(random_numbers, dev_random_numbers,
                              n_random_numbers * sizeof(double),
                              hipMemcpyDeviceToHost));
    }


double RngGpu::next_rand(const double mean, const double dev) {
    if (index_random_numbers >= n_random_numbers) {
        index_random_numbers = 0;
        hipDeviceSynchronize();
        if (gaussian) {
            CURAND_CALL(hiprandGenerateNormalDouble(gen, dev_random_numbers,
                                                   n_random_numbers, 0.0, 1.0));
        } else {
            CURAND_CALL(hiprandGenerateUniformDouble(gen, dev_random_numbers,
                                                    n_random_numbers));
        }
        CUDA_CALL(hipMemcpyAsync(random_numbers, dev_random_numbers,
                                  n_random_numbers * sizeof(double),
                                  hipMemcpyDeviceToHost));
        }
    return random_numbers[index_random_numbers++] * dev + mean;
    }

RngGpu::~RngGpu() {
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(dev_random_numbers));
    CUDA_CALL(hipHostFree(random_numbers));
    }
