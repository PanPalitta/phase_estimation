#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <map>
#include <vector>

#include "rng_gpu.h"

using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
     cout << "CUDA call error at" << __FILE__<< ":" << __LINE__ << endl;\
     }} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
     cout << "CURAND call error at" << __FILE__<< ":" << __LINE__ << endl;\
     exit(-1);\
     }} while(0)

/// Note that this function was lifted from http://code.google.com/p/gpmr/
void setDevice(int commRank, int commSize)
{
    int devCount;
    int deviceNum=0;
    CUDA_CALL(hipGetDeviceCount(&devCount));
    FILE * fp = popen("/bin/hostname", "r");
    char buf[1024];
    if (fgets(buf, 1023, fp) == NULL) strcpy(buf, "localhost");
    pclose(fp);
    string host = buf;
    host = host.substr(0, host.size() - 1);
    strcpy(buf, host.c_str());
    if (commRank == 0)
    {
        map<string, vector<int> > hosts;
        map<string, int> devCounts;
        hosts[buf].push_back(0);
        devCounts[buf] = devCount;

        MPI_Status stat;
        MPI_Request req;
        for (int i = 1; i < commSize; ++i)
        {
            MPI_Recv(buf, 1024, MPI_CHAR, i, 0, MPI_COMM_WORLD, &stat);
            MPI_Recv(&devCount, 1, MPI_INT, i, 0, MPI_COMM_WORLD, &stat);

            // check to make sure each process on each node reports the same number of devices.
            hosts[buf].push_back(i);
            if (devCounts.find(buf) != devCounts.end())
            {
                if (devCounts[buf] != devCount)
                {
                    printf("Error, device count mismatch %d != %d on %s\n", devCounts[buf], devCount, buf);
                    fflush(stdout);
                }
            }
            else devCounts[buf] = devCount;
        }
        // check to make sure that we don't have more jobs on a node than we have GPUs.
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it)
        {
            if (it->second.size() > static_cast<unsigned int>(devCounts[it->first]))
            {
                printf("Error, more jobs running on '%s' than devices - %d jobs > %d devices.\n",
                       it->first.c_str(), static_cast<int>(it->second.size()), devCounts[it->first]);
                fflush(stdout);
                exit(1);
            }
        }

        // send out the device number for each process to use.
        MPI_Irecv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &req);
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it)
        {
            for (unsigned int i = 0; i < it->second.size(); ++i)
            {
                int devID = i;
                MPI_Send(&devID, 1, MPI_INT, it->second[i], 0, MPI_COMM_WORLD);
            }
        }
        MPI_Wait(&req, &stat);
    }
    else
    {
        // send out the hostname and device count for your local node, then get back the device number you should use.
        MPI_Status stat;
        MPI_Send(buf, strlen(buf) + 1, MPI_CHAR, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&devCount, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
        MPI_Recv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &stat);
    }
    MPI_Barrier(MPI_COMM_WORLD);
    CUDA_CALL(hipSetDevice(deviceNum));
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    cout <<  device << " " << devProp.name << " Compute Capability: " << devProp.major << "." << devProp.minor << "\n";
}

void RngGpu::RngGpu(int n_urandom_numbers, int n_grandom_numbers):
    n_urandom_numbers(_n_urandom_numbers),
    n_grandom_numbers(_n_urandom_numbers) {

    urandom_numbers = new double[n_urandom_numbers];
    index_urandom_numbers = 0;
    grandom_numbers = new double[n_grandom_numbers];
    index_grandom_numbers = 0;
    CUDA_CALL(hipMalloc((void **)&dev_urandom_numbers,
                         n_urandom_numbers*sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&dev_grandom_numbers,
                         n_grandom_numbers*sizeof(double)));
    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen,
                                      HIPRAND_RNG_PSEUDO_DEFAULT));
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,
                1234ULL));
    CURAND_CALL(hiprandGenerateUniformDouble(gen, dev_urandom_numbers,
                                            n_urandom_numbers));
    CURAND_CALL(hiprandGenerateNormalDouble(gen, dev_grandom_numbers,
                                           n_grandom_numbers, 0.0, 1.0));
    CUDA_CALL(hipMemcpy(urandom_numbers, dev_urandom_numbers,
                         n_urandom_numbers * sizeof(double),
                         hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(grandom_numbers, dev_grandom_numbers,
                         n_grandom_numbers * sizeof(double),
                         hipMemcpyDeviceToHost));
}

double RngVsl::next_grand(const double mean, const double dev) {
    if index_grandom_numbers >= n_grandom_numbers {
        index_grandom_numbers = 0;
        CUDA_CALL(hipMemcpy(grandom_numbers, dev_grandom_numbers,
                             n_grandom_numbers * sizeof(double),
                             hipMemcpyDeviceToHost));
        CURAND_CALL(hiprandGenerateNormalDouble(gen, dev_grandom_numbers,
                                               n_grandom_numbers, 0.0, 1.0));
    }
  	return grandom_numbers[index_grandom_numbers++]*dev+mean;
}

double RngVsl::next_grand(const double mean, const double dev) {
    if index_urandom_numbers >= n_urandom_numbers {
        index_urandom_numbers = 0;
        CUDA_CALL(hipMemcpy(urandom_numbers, dev_urandom_numbers,
                             n_urandom_numbers * sizeof(double),
                             hipMemcpyDeviceToHost));
        CURAND_CALL(hiprandGenerateUniformDouble(gen, dev_urandom_numbers,
                                                n_urandom_numbers));
    }
  	return urandom_numbers[index_urandom_numbers++];
}


void RngGpu::~RngGpu() {
    delete[] grandom_numbers;
    delete[] urandom_numbers;
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(dev_urandom_numbers));
    CUDA_CALL(hipFree(dev_grandom_numbers));
}
