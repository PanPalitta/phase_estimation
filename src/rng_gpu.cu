#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "rng_gpu.h"

using namespace std;

double *dev_urandom_numbers;
double *dev_grandom_numbers;
hiprandGenerator_t gen;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
     cout << "CUDA call error at" << __FILE__<< ":" << __LINE__ << endl;\
     }} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
     cout << "CURAND call error at" << __FILE__<< ":" << __LINE__ << endl;\
     }} while(0)

void gpu_cache_alloc(int n_urandom_numbers, int n_grandom_numbers) {
    CUDA_CALL(hipMalloc((void **)&dev_urandom_numbers,
              n_urandom_numbers*sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&dev_grandom_numbers,
              n_grandom_numbers*sizeof(double)));
    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT));
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,
                1234ULL));
}

void gpu_cache_init(double *urandom_numbers, int n_urandom_numbers, 
                    double *grandom_numbers, int n_grandom_numbers) {
  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniformDouble(gen, dev_urandom_numbers, 
                                          n_urandom_numbers));
  CUDA_CALL(hipMemcpy(urandom_numbers, dev_urandom_numbers, 
                       n_urandom_numbers * sizeof(double),
                       hipMemcpyDeviceToHost));
  CURAND_CALL(hiprandGenerateNormalDouble(gen, dev_grandom_numbers, 
                                         n_grandom_numbers, 0.0, 1.0));
  CUDA_CALL(hipMemcpy(grandom_numbers, dev_grandom_numbers, 
                       n_grandom_numbers * sizeof(double),
                       hipMemcpyDeviceToHost));
}

void gpu_cache_free() {
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(dev_urandom_numbers));
    CUDA_CALL(hipFree(dev_grandom_numbers));
}
