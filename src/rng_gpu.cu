#include <iostream>
#include <map>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "rng_gpu.h"

using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
            cout << "CUDA call error at" << __FILE__<< ":" << __LINE__ << endl;\
            }} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
            cout << "CURAND call error at" << __FILE__<< ":" << __LINE__ << endl;\
            exit(-1);\
            }} while(0)

/// Note that this function was lifted from http://code.google.com/p/gpmr/
void setDevice(int commRank, int commSize) {
    int devCount;
    int deviceNum = 0;
    CUDA_CALL(hipGetDeviceCount(&devCount));
    FILE * fp = popen("/bin/hostname", "r");
    char buf[1024];
    if (fgets(buf, 1023, fp) == NULL) strcpy(buf, "localhost");
    pclose(fp);
    string host = buf;
    host = host.substr(0, host.size() - 1);
    strcpy(buf, host.c_str());
    if (commRank == 0) {
        map<string, vector<int> > hosts;
        map<string, int> devCounts;
        hosts[buf].push_back(0);
        devCounts[buf] = devCount;

        MPI_Status stat;
        MPI_Request req;
        for (int i = 1; i < commSize; ++i) {
            MPI_Recv(buf, 1024, MPI_CHAR, i, 0, MPI_COMM_WORLD, &stat);
            MPI_Recv(&devCount, 1, MPI_INT, i, 0, MPI_COMM_WORLD, &stat);

            // check to make sure each process on each node reports the same number of devices.
            hosts[buf].push_back(i);
            if (devCounts.find(buf) != devCounts.end()) {
                if (devCounts[buf] != devCount) {
                    printf("Error, device count mismatch %d != %d on %s\n", devCounts[buf], devCount, buf);
                    fflush(stdout);
                    }
                }
            else devCounts[buf] = devCount;
            }
        // check to make sure that we don't have more jobs on a node than we have GPUs.
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            if (it->second.size() > static_cast<unsigned int>(devCounts[it->first])) {
                printf("Error, more jobs running on '%s' than devices - %d jobs > %d devices.\n",
                       it->first.c_str(), static_cast<int>(it->second.size()), devCounts[it->first]);
                fflush(stdout);
                exit(1);
                }
            }

        // send out the device number for each process to use.
        MPI_Irecv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &req);
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            for (unsigned int i = 0; i < it->second.size(); ++i) {
                int devID = i;
                MPI_Send(&devID, 1, MPI_INT, it->second[i], 0, MPI_COMM_WORLD);
                }
            }
        MPI_Wait(&req, &stat);
        }
    else {
        // send out the hostname and device count for your local node, then get back the device number you should use.
        MPI_Status stat;
        MPI_Send(buf, strlen(buf) + 1, MPI_CHAR, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&devCount, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
        MPI_Recv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &stat);
        }
    MPI_Barrier(MPI_COMM_WORLD);
    CUDA_CALL(hipSetDevice(deviceNum));
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    cout <<  device << " " << devProp.name << " Compute Capability: " << devProp.major << "." << devProp.minor << "\n";
    }

RngGpu::RngGpu(int _n_urandom_numbers, int _n_grandom_numbers):
    n_urandom_numbers(_n_urandom_numbers),
    n_grandom_numbers(_n_grandom_numbers) {

    CUDA_CALL(hipHostMalloc((void **)&urandom_numbers,
                             n_urandom_numbers * sizeof(double)));
    CUDA_CALL(hipHostMalloc((void **)&grandom_numbers,
                             n_grandom_numbers * sizeof(double)));
    index_urandom_numbers = n_urandom_numbers;
    index_grandom_numbers = n_grandom_numbers;
    CUDA_CALL(hipMalloc((void **)&dev_urandom_numbers,
                         n_urandom_numbers * sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&dev_grandom_numbers,
                         n_grandom_numbers * sizeof(double)));
    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    CURAND_CALL(hiprandGenerateUniformDouble(gen, dev_urandom_numbers,
                                            n_urandom_numbers));
    CURAND_CALL(hiprandGenerateNormalDouble(gen, dev_grandom_numbers,
                                           n_grandom_numbers, 0.0, 1.0));
    CUDA_CALL(hipMemcpyAsync(urandom_numbers, dev_urandom_numbers,
                              n_urandom_numbers * sizeof(double),
                              hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpyAsync(grandom_numbers, dev_grandom_numbers,
                              n_grandom_numbers * sizeof(double),
                              hipMemcpyDeviceToHost));
    }

double RngGpu::next_urand() {
    if (index_urandom_numbers >= n_urandom_numbers) {
        index_urandom_numbers = 0;
        hipDeviceSynchronize();
        CURAND_CALL(hiprandGenerateUniformDouble(gen, dev_urandom_numbers,
                                                n_urandom_numbers));
        CUDA_CALL(hipMemcpyAsync(urandom_numbers, dev_urandom_numbers,
                                  n_urandom_numbers * sizeof(double),
                                  hipMemcpyDeviceToHost));
        }
    return urandom_numbers[index_urandom_numbers++];
    }

double RngGpu::next_grand(const double mean, const double dev) {
    if (index_grandom_numbers >= n_grandom_numbers) {
        index_grandom_numbers = 0;
        hipDeviceSynchronize();
        CURAND_CALL(hiprandGenerateNormalDouble(gen, dev_grandom_numbers,
                                               n_grandom_numbers, 0.0, 1.0));
        CUDA_CALL(hipMemcpyAsync(grandom_numbers, dev_grandom_numbers,
                                  n_grandom_numbers * sizeof(double),
                                  hipMemcpyDeviceToHost));
        }
    return grandom_numbers[index_grandom_numbers++] * dev + mean;
    }

RngGpu::~RngGpu() {
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(dev_urandom_numbers));
    CUDA_CALL(hipFree(dev_grandom_numbers));
    CUDA_CALL(hipHostFree(grandom_numbers));
    CUDA_CALL(hipHostFree(urandom_numbers));
    }
